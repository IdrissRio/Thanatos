#include "hip/hip_runtime.h"
 //
//  Client_ImplementationGPU.c
//  Server
//
//  Created by IdrissRio on 26/05/16.
//  Copyright © 2016 RIFT. All rights reserved.
//

#include "Client_ImplementationGPU.cuh"
#include "mpi.h"
#include "omp.h"




__device__ char mess_known[] = {'i', 'l', ' ', 'm', 'e', 's', 's', 'a', 'g', 'g', 'i', 'o', ' ', 's', 'e', 'g', 'r', 'e', 't', 'o', ' ', 'e', 39, ':', 0};


__host__ __device__ unsigned char get_bit_long_cu(byte *in, long long pos){
    int  pos_byte = pos / 8;
    short pos_bit = pos % 8;
    byte selected_byte = in[pos_byte];
    return ((selected_byte >> (7 - pos_bit)) & 0x1);
}

__device__ void set_bit_long_cu(byte *in, long long pos, short val) {
    int  pos_byte = pos / 8;
    short pos_bit = pos % 8;
    byte selected_byte = in[pos_byte];
    selected_byte = (byte)(((0xFF7F >> pos_bit) & selected_byte) & 0x00FF);
    byte new_byte = (byte)((val << (7 - pos_bit)) | selected_byte);
    in[pos_byte] = new_byte;
    return;
}

__device__ unsigned char get_bit_cu(byte *in, int pos){
    int pos_byte = pos / 8;
    short pos_bit = pos % 8;
    byte selected_byte = in[pos_byte];
    return ((selected_byte >> (7 - pos_bit)) & 0x1);
}

__device__ void set_bit_cu(byte *in, int pos, short val) {
    int pos_byte = pos / 8;
    short pos_bit = pos % 8;
    byte selected_byte = in[pos_byte];
    selected_byte = (byte)(((0xFF7F >> pos_bit) & selected_byte) & 0x00FF);
    byte new_byte = (byte)((val << (7 - pos_bit)) | selected_byte);
    in[pos_byte] = new_byte;
    return;
}


__global__ void generate_all_subkeys_cu(long long offset, byte *key, byte *sub_key, byte *c, byte *d, byte *cd, byte *copy){
    int tid = (blockIdx.x + gridDim.x * blockIdx.y) * blockDim.x + threadIdx.x;
    long long key_in_block = offset + tid;
    for (short i = 8; i < 64; ++i){
        short val = get_bit_from_longlong_cu(key_in_block, i);
        set_bit_cu(key, (tid * 7 * 8) + i - 8, val);
    }
    const short SHIFTS_LEN = 16;
    //select_bits_with_pos_in_key(*the_key, 0, PC1_LEN/2, &c);
    //byte copy[8];
    for (short i = 0; i < 7; ++i)
        copy[tid*8 + i] = key[(tid * 7) + i];
    
    for (short i = 0; i < 28; ++i){
        unsigned char val = get_bit_cu(copy, tid*8*8 + i);
        set_bit_cu(c, (tid * 4 * 8) + i, val);
    }
    //select_bits_with_pos_in_key(*the_key, PC1_LEN/2, PC1_LEN/2, &d);
    for (short i = 28; i < 56; ++i){
        unsigned char val = get_bit_cu(copy, tid*8*8 + i);
        set_bit_cu(d, (tid * 4 * 8) + i - 28, val);
    }
    
    for (short i = 0; i < 4; ++i)
        copy[tid*8 + i] = c[(tid * 4) + i];
    for (short i = 4; i < 8; ++i)
        copy[tid*8 + i] = d[(tid * 4) + i - 4];
    for (short cycle = 0; cycle < SHIFTS_LEN; ++cycle){
        //rotate_left(&c, 28, SHIFTS_cu[i]);
        for (short i = 0; i < 28; ++i) {
            short val = get_bit_cu(copy,tid*8*8 + (((i + SHIFTS_cu[cycle]) % 28)));
            set_bit_cu(c, (tid * 4 * 8) + i, val);
        }
        // rotate_left(&d, 28, SHIFTS[i]);
        for (short i = 32; i < 60; ++i) {
            short val = get_bit_cu(copy, tid*8*8 + (((i - 32 + SHIFTS_cu[cycle]) % 28)) + 32);
            set_bit_cu(d, (tid * 4 * 8) + i - 32, val);
        }
        
        for (short i = 0; i < 4; ++i)
            copy[tid*8 + i] = c[(tid * 4) + i];
        for (short i = 4; i < 8; ++i)
            copy[tid*8 + i] = d[(tid * 4) + i - 4];
        
        //concatenate_bits(c, 28, d, 28, &cd);
        int j = 0;
        for (int i = 0; i < 28; ++i){
            unsigned char val = get_bit_cu(c, (tid * 4 * 8) + i);
            set_bit_cu(cd, (tid * 7 * 8) + j, val);
            ++j;
        }
        
        for (int i = 0; i < 28; ++i){
            unsigned char val = get_bit_cu(d, (tid * 4 * 8) + i);
            set_bit_cu(cd, (tid * 7 * 8) + j, val);
            ++j;
        }
        //select_bits(&cd, PC2_cu, 48);
        for (short i = 0; i < 48; ++i){
            unsigned char val = get_bit_cu(cd, (tid * 7 * 8) + (PC2_cu[i] - 1));
            set_bit_long_cu(sub_key, (long long)(tid * 96LL * 8LL) + (long long)(cycle * 48LL) + (long long)i, val);
        }
        
    }
    
}







__device__ short get_bit_from_longlong_cu(long long in, short pos){
    return ((in >> (63 - pos)) & 0x1);
}



__global__ void try_all_cu(byte *sub_keys, byte *crypted_msg, byte *cipher_decrypted,
                           byte *left, byte *right, byte *left_right,
                           byte *r_backup, byte *r_commuted, long long *res, byte *copy_8 ){
    int tid = (blockIdx.x + gridDim.x * blockIdx.y) * blockDim.x + threadIdx.x;
    unsigned char partial = 0;
    
    while (partial < 3){
        /******************************* cipher_decrypt *****************************/
        for (short i = 0; i < 64; ++i){
            unsigned char val = get_bit_cu(crypted_msg, (partial * 64) + (IP_cu[i] - 1));
            set_bit_cu(cipher_decrypted, (tid*64) + i, val);
        }
        
        for (short i = 0; i < 32; ++i){
            unsigned char val = get_bit_cu(cipher_decrypted, (tid * 64) + i);
            set_bit_cu(left, (tid * 32) + i, val);
        }
        
        for (short i = 32; i < 64; ++i){
            unsigned char val = get_bit_cu(cipher_decrypted, (tid * 64) + i);
            set_bit_cu(right, (tid*32) + i - 32, val);
        }
        for (short key_counter = 0; key_counter < 16; ++key_counter){
            for (short copy_counter = 0; copy_counter < 4; ++copy_counter)
                r_backup[(tid * 4) + copy_counter] = right[(tid * 4) + copy_counter];
            for (short i = 0; i < 48; ++i){
                unsigned char val = get_bit_cu(right, (tid * 32) + (E_cu[i] - 1));
                set_bit_cu(r_commuted, (tid * 48) + i, val);
            }
            for (short i = 0; i < 6; ++i)
                copy_8[tid*8 + i] = r_commuted[(tid*6)+i];
            for (short i = 0; i < 6; ++i)
                r_commuted[(tid * 6)+i] = copy_8[tid*8 + i] ^ sub_keys[tid * 96 + 6 * (15 - key_counter) + i];
            for (short i = 0; i < 8; ++i){
                for (short j = 0; j < 6; ++j){
                    short val = get_bit_cu(r_commuted, (tid * 48)+(6 * i + j));
                    set_bit_cu(copy_8,tid*8*8 +  8 * i + j, val);
                }
            }
            
            int lh_byte = 0;
            for (short b = 0; b < 8; ++b) { // Should be sub-blocks
                byte val_byte = copy_8[tid*8 + b];
                short r = 2 * ((val_byte >> 7) & 0x0001) + ((val_byte >> 2) & 0x0001); // 1 and 6
                short c = (val_byte >> 3) & 0x000F; // Middle 4 bits
                short h_byte = S_cu[(64 * b) + (16 * r) + c]; // 4 bits (half byte) output
                if (b % 2 == 0) lh_byte = h_byte; // Left half byte
                else right[(tid * 4) + (b / 2)] = (byte)(16 * lh_byte + h_byte);
                /* * * * * * * * * * * * * * * * * * * * * * */
            }
            
            //select_bits(&right, P, 32);
            for (short i = 0; i < 32; ++i){
                unsigned char val = get_bit_cu(right, (tid * 32) + P_cu[i] - 1);
                set_bit_cu(copy_8, tid*8*8 + i, val);
            }
            for (short i = 0; i < 4; ++i)
                right[(tid * 4) + i] = copy_8[tid*8 + i];
            for (short i = 0; i < 4; ++i)
                right[(tid * 4) + i] = copy_8[tid*8 + i] ^ left[(tid * 4) + i];
            for (int i = 0; i < 4; ++i)
                left[(tid * 4) + i] = r_backup[(tid * 4) + i];
        }
        
        short j = 0;
        for (int i = 0; i < 32; ++i){
            unsigned char val = get_bit_cu(right, (tid * 32) + i);
            set_bit_cu(left_right, (tid * 64) + j, val);
            ++j;
        }
        for (int i = 0; i < 32; ++i){
            unsigned char val = get_bit_cu(left, (tid * 32) + i);
            set_bit_cu(left_right, (tid * 64) + j, val);
            ++j;
        }
        
        for (short i = 0; i < 64; ++i){
            unsigned char val = get_bit_cu(left_right, (tid * 64) + INVP_cu[i] - 1);
            set_bit_cu(cipher_decrypted, (tid * 64) + i, val);
        }
        for (short i = 0; i < 8; ++i)
            if (cipher_decrypted[(tid * 8) + i] != mess_known[(partial * 8) + i]){
                return;
            }
        ++partial;
        
    }
    *res = tid;
    return;
}

//##################################################################################################
//##################################################################################################
//##################################################################################################
//##################################################################################################
//##################################################################################################
//##################################################################################################

extern "C"  void testOffsetGpu(byte **MODIFIED_MESSAGE, byte *SUCCESS,int processID){

int block[4]={1,1,1,1};
MPI_Datatype types[4]={MPI_LONG_LONG,MPI_LONG_LONG,MPI_DOUBLE,MPI_INT};
MPI_Aint disallineamentoBIT[4]={offsetof(infoExec,possibileChiave),offsetof(infoExec,processID),offsetof(infoExec,tempo),offsetof(infoExec,tipo)};
MPI_Type_struct(4, block, disallineamentoBIT, types, &Info_Type);
MPI_Type_commit(&Info_Type);
infoExec informazioni;
informazioni.tipo=0;
    short thread_number = 256 ;
    long long block_size = 16777216;
    long long cmc = -1;
    int num_gpus = 0;
    int exit_Condition=0;
    hipGetDeviceCount(&num_gpus);
    omp_set_num_threads(num_gpus);
    //Calcoliamo il tempo
    informazioni.processID=processID;
    informazioni.possibileChiave=block_size;
//Calcoliamo il tempo

#pragma omp parallel
{

float tmp=0.0;
     	     byte *sub_keys, *key, *c, *d, *cd, *cipher_decrypted, *copy, *r_backup, *r_commuted, *left_right,*crypted_msg;
            long long *res;

            short tid = omp_get_thread_num();
            hipSetDevice(tid);
            hipMalloc((void **)&res, sizeof(long long));
            hipMalloc((void **)&sub_keys, block_size * sizeof(byte) * 96);
            hipMalloc((void **)&key, block_size * sizeof(byte) * 7);
            hipMalloc((void **)&c, block_size * sizeof(byte) * 4);
            hipMalloc((void **)&d, block_size * sizeof(byte) * 4);
            hipMalloc((void **)&cd, block_size * sizeof(byte) * 7);
            hipMalloc((void **)&copy, block_size * sizeof(byte) * 8);
            hipMalloc((void **)&crypted_msg, sizeof(byte) * 24);
            hipMemcpy(crypted_msg, *MODIFIED_MESSAGE, sizeof(byte)*24, hipMemcpyHostToDevice);
            hipMemcpy(res, &cmc, sizeof(long long), hipMemcpyHostToDevice);
            hipMalloc((void **)&cipher_decrypted, block_size * sizeof(byte) * 8);
            hipMalloc((void **)&r_backup, block_size * sizeof(byte) * 4);
            hipMalloc((void **)&r_commuted, block_size * sizeof(byte) * 6);
            hipMalloc((void **)&left_right, block_size * sizeof(byte) * 8);
        
           long long d_offset;
            while(cmc == -1 && exit_Condition == 0 ){
#pragma omp critical  //Sezione Critica
                {
                    MPI_Send(&informazioni, sizeof(Info_Type), Info_Type, 0, 1, MPI_COMM_WORLD);
                    MPI_Recv(&d_offset, 1, MPI_LONG_LONG, 0, 3, MPI_COMM_WORLD, MPI_STATUS_IGNORE); //Riceviamo il nuovo offset e lo testiamo
		if(d_offset==-1){
		       exit_Condition=-1;
		       d_offset=0;
		    }
                }
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start);
                generate_all_subkeys_cu<<<dim3(thread_number, thread_number), thread_number>>>(d_offset, key, sub_keys, c, d, cd, copy);
                try_all_cu<<<dim3(thread_number, thread_number), thread_number>>>(sub_keys,crypted_msg,cipher_decrypted, c, d, left_right,r_backup,r_commuted,res, copy);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&tmp,start,stop);
                hipMemcpy(&cmc, res, sizeof(long long), hipMemcpyDeviceToHost);
                if (cmc != -1){
                    informazioni.possibileChiave=d_offset+cmc;
                    informazioni.tempo=tmp/1000;
                    MPI_Send(&informazioni, sizeof(Info_Type), Info_Type, 0, 0, MPI_COMM_WORLD); //Inviamo al server la chiave
                }
        informazioni.tempo=tmp/1000;


}
            hipFree(key);
            hipFree(c);
            hipFree(d);
            hipFree(cd);
            hipFree(left_right);
            hipFree(r_commuted);
            hipFree(r_backup);
            hipFree(cipher_decrypted);
            hipFree(crypted_msg);
            hipFree(sub_keys);
            hipFree(res);
            hipFree(copy);
        
    }
}




extern "C"  void testOffsetGpuJetson(byte *MODIFIED_MESSAGE, byte *SUCCESS,int processID){


int block[4]={1,1,1,1};
MPI_Datatype types[4]={MPI_LONG_LONG,MPI_LONG_LONG,MPI_DOUBLE,MPI_INT};
MPI_Aint disallineamentoBIT[4]={offsetof(infoExec,possibileChiave),offsetof(infoExec,processID),offsetof(infoExec,tempo),offsetof(infoExec,tipo)};
MPI_Type_struct(4, block, disallineamentoBIT, types, &Info_Type);
MPI_Type_commit(&Info_Type);
infoExec informazioni;
informazioni.tipo=0;

short thread_number = 256;
long long block_size = 16777216;
hipSetDeviceFlags(hipDeviceMapHost);
byte *sub_keys, *key, *c, *d, *cd, *cipher_decrypted, *r_backup, *r_commuted, *left_right, *crypted_msg, *copy;
byte *d_sub_keys, *d_key, *d_c, *d_d, *d_cd, *d_cipher_decrypted, *d_r_backup, *d_r_commuted, *d_left_right, *d_crypted_msg, *d_copy;
long long d_offset = 0;
long long *res;
long long *cmc;
hipHostAlloc((void **)&cmc, sizeof(long long), hipHostMallocMapped);
*cmc = -1;

informazioni.processID=processID;
informazioni.possibileChiave=block_size;

hipHostGetDevicePointer((void **)&res, (void *)cmc, 0);

hipHostAlloc((void **)&copy, block_size * sizeof(byte) * 8, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_copy, (void *)copy, 0);

hipHostAlloc((void **)&sub_keys, block_size * sizeof(byte) * 96, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_sub_keys, (void *)sub_keys, 0);

hipHostAlloc((void **)&key, block_size * sizeof(byte) * 7, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_key, (void *)key, 0);

hipHostAlloc((void **)&c, block_size * sizeof(byte) * 4, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_c, (void *)c, 0);

hipHostAlloc((void **)&d, block_size * sizeof(byte) * 4, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_d, (void *)d, 0);

hipHostAlloc((void **)&cd, block_size * sizeof(byte) * 7, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_cd, (void *)cd, 0);
hipHostAlloc((void **)&crypted_msg, sizeof(byte) * 24, hipHostMallocMapped);
for (short i = 0; i<24; ++i){
crypted_msg[i] = MODIFIED_MESSAGE[i];
}
hipHostGetDevicePointer((void **)&d_crypted_msg, (void *)crypted_msg, 0);
hipHostAlloc((void **)&cipher_decrypted, block_size * sizeof(byte) * 8, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_cipher_decrypted, (void *)cipher_decrypted, 0);
hipHostAlloc((void **)&r_backup, block_size * sizeof(byte) * 4, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_r_backup, (void *)r_backup, 0);
hipHostAlloc((void **)&r_commuted, block_size * sizeof(byte) * 6, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_r_commuted, (void *)r_commuted, 0);
hipHostAlloc((void **)&left_right, block_size * sizeof(byte) * 8, hipHostMallocMapped);
hipHostGetDevicePointer((void **)&d_left_right, (void *)left_right, 0);

//Calcoliamo il tempo
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);


while(true){
    MPI_Send(&informazioni, sizeof(Info_Type), Info_Type, 0, 1, MPI_COMM_WORLD);//Comunichiamo al server che abbiamo finito la parte scatter e serve un nuovo offset da analizzare.
    MPI_Recv(&d_offset, 1, MPI_LONG_LONG, 0, 3, MPI_COMM_WORLD, MPI_STATUS_IGNORE); //Riceviamo il nuovo offset e lo testiamo
    if(d_offset==-1){
        break;
    }
    hipEventRecord(start);
    generate_all_subkeys_cu << <dim3(thread_number, thread_number), thread_number >> >(d_offset, d_key, d_sub_keys, d_c, d_d, d_cd, d_copy);
    hipDeviceSynchronize();
    try_all_cu << <dim3(thread_number, thread_number), thread_number >> >(d_sub_keys, d_crypted_msg, d_cipher_decrypted, d_c, d_d, d_left_right, d_r_backup, d_r_commuted, res, d_copy);
    hipDeviceSynchronize();
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float tmp=0.0;
    hipEventElapsedTime(&tmp,start,stop);
informazioni.tempo=tmp/1000.0;
    if (*cmc != -1){
        informazioni.possibileChiave=d_offset+ *cmc;
        MPI_Send(&informazioni, sizeof(Info_Type), Info_Type, 0, 0, MPI_COMM_WORLD); //Inviamo al server la chiave
    break;
    }
}




hipHostFree(key);
hipHostFree(c);
hipHostFree(d);
hipHostFree(cd);

hipHostFree(left_right);
hipHostFree(r_commuted);
hipHostFree(r_backup);
hipHostFree(cipher_decrypted);
hipHostFree(sub_keys);
hipHostFree(res);
hipHostFree(crypted_msg);
hipHostFree(copy);
return;
}


